// todo: - remove dependence on model matrix, it can't be used anymore because the basis for animated / deformed surfaces is a lattice, not a matrix
//       - implement dynamic render targets creation / support
//       - implement optimized surface reconstruction shader using push -> pull, do low frequency indirect lighting of generated micro directional lights at low res reconstruction
//         do directional screen space occlusion for high frequency indirect lighting at half res
//       - provide API interface for attaching shadowmaps to lights
//       - implement shadow mapping with soft shadowing filters, in the lighting step add in the low frequency and high frequency indirect contributions
//       - implement hdr postprocessing (tone mapping, bloom, depth of field, motion blur)
//       - provide API interface for mixed (traditional / axel) rendering
//       - implement soft particles rendering for smoke / fog / fire
//       - take a look at the possibility of allowing core to render a layer of transparent surfaces (for water mostly) (meh priority)

#include "vtrace.h"

//#ifdef _DEBUG

#define CheckCgError(function) __CheckCgError(function, __FILE__, __LINE__)
#define AbortCgError(function) __AbortCgError(function, __FILE__, __LINE__)

#define SafeRelease(pointer) if(pointer) {(pointer)->Release(); (pointer) = 0;}
/*
#else

#define CheckCgError(function) false
#define AbortCgError(function) ;

#define SafeRelease(pointer) {(pointer)->Release(); (pointer) = 0;}

#endif*/

inline bool __CheckCgError(const std::string &function, const char *file, const int line)
{
	CGerror error; const char * errorMessage(cgGetLastErrorString(&error));

	if (error == CG_NO_ERROR)
	{
		return false;
	}

	std::cerr << "Error calling " << function << "() at " << file << ":" << line << " - " << errorMessage << std::endl;

	if (error == CG_COMPILER_ERROR)
	{
		std::cerr << cgGetLastListing(__engine->g_context) << std::endl;
	}

	return true;
}

inline void __AbortCgError(const std::string &function, const char *file, const int line)
{
	if (__CheckCgError(function, file, line))
	{
		//exit(-1);
	}
}

void vEngine::LightDirectional(const vVec3 *direction, const vVec3 *diffuse, const vVec3 *specular)
{
	vMat4x4 matModelView;
	vVec3 vsOrigin(0,0,0), vsDirection;

	vMatrixMultiply(&matModelView, &viewMatrix, &modelMatrix);
	vMatrixProjectVec3(&vsOrigin, &matModelView, &vsOrigin);
	vMatrixProjectVec3(&vsDirection, &matModelView, direction);

	AddLight(new DirectionalLight(vsDirection-vsOrigin, *diffuse, *specular));
}

void vEngine::LightPoint(const vVec3 *position, const vVec3 *diffuse, const vVec3 *specular)
{
	vMat4x4 matModelView;
	vVec3	vsPosition;

	vMatrixMultiply(&matModelView, &viewMatrix, &modelMatrix);
	vMatrixProjectVec3(&vsPosition, &matModelView, position);

	AddLight(new PointLight(vsPosition, *diffuse, *specular));
}

int vEngine::CompositeInit(bool vsync, bool windowed, bool tripleBuffering)
{
	view = 1;

	bDeviceLost = false;
	bWindowed   = windowed;

	for(int i = 0; i < NUM_PASSES; i++)
	{
		passEnabled[i] = true;
	}

	pD3D = Direct3DCreate9(D3D_SDK_VERSION);

	if (pD3D == NULL)
	{
		return -1;
	}
   
	unsigned int adapterNum;

	for (adapterNum = 0; adapterNum < pD3D->GetAdapterCount(); adapterNum++)
	{
		D3DADAPTER_IDENTIFIER9 adapterId;
		pD3D->GetAdapterIdentifier(adapterNum, 0, &adapterId);

		int deviceNum;
		cudaD3D9GetDevice(&deviceNum, adapterId.DeviceName);

		if (hipGetLastError() == hipSuccess) 
		{
			break;
		}
	}

	if (adapterNum == pD3D->GetAdapterCount())
	{
		printf("Unable to find CUDA->D3D9 capable device\n");

		return -1;
	}

	RECT              rc; GetClientRect(hWindow,&rc);
	D3DDISPLAYMODE d3ddm; pD3D->GetAdapterDisplayMode(D3DADAPTER_DEFAULT, &d3ddm);

	D3DPRESENT_PARAMETERS d3dpp; ZeroMemory(&d3dpp, sizeof(d3dpp));

	d3dpp.Windowed               = bWindowed;
	d3dpp.BackBufferCount        = 1 + tripleBuffering;
	d3dpp.SwapEffect             = D3DSWAPEFFECT_DISCARD;
	d3dpp.hDeviceWindow          = hWindow;
	d3dpp.BackBufferWidth	     = rc.right  - rc.left;
	d3dpp.BackBufferHeight       = rc.bottom - rc.top;
	d3dpp.BackBufferFormat       = d3ddm.Format;

	if (!vsync)
	{
		d3dpp.PresentationInterval = D3DPRESENT_INTERVAL_IMMEDIATE;
	}

	if (FAILED(pD3D->CreateDevice(D3DADAPTER_DEFAULT, D3DDEVTYPE_HAL, hWindow, D3DCREATE_PUREDEVICE | D3DCREATE_HARDWARE_VERTEXPROCESSING, &d3dpp, &pD3DDevice)))
	{
		return -1;
	}

	pD3DDevice->GetRenderTarget(0, &frameBuffer);

	cudaD3D9SetDirect3DDevice(pD3DDevice);
	cutilCheckMsg("cudaD3D9SetDirect3DDevice failed");

	g_context = context = cgCreateContext();
	AbortCgError("cgCreateContext");

	cgD3D9SetDevice(pD3DDevice);
	AbortCgError("cgD3D9SetDevice");

	cgD3D9RegisterStates(context);
	AbortCgError("cgD3D9RegisterStates");

	cgD3D9SetManageTextureParameters(context, CG_TRUE);
	AbortCgError("cgD3D9SetManageTextureParameters");

	if (FAILED(InitDeviceObjects()))
	{
		return -1;
	}

	return 0;
}

int vEngine::CompositeShutdown()
{
	for (EffectMap::iterator it(effects.begin()); it!=effects.end(); ++it)
	{
		CGeffect effect(it->second);

		if (effect)
		{
			cgDestroyEffect(effect);
		}
	}

	cgD3D9SetDevice(0);
	cgDestroyContext(context);

	DestroyDeviceObjects();

	SafeRelease(pD3DDevice);
	SafeRelease(pD3D);

	return 0;
}

int vEngine::CompositeCreateRenderSurface(CompositeRenderSurface * renderSurface, unsigned int width, unsigned int height, unsigned int flags)
{
	// TODO: Create the composite render surface

	return -1;
}

int vEngine::CompositeFreeRenderSurface(CompositeRenderSurface * renderSurface)
{
	// TODO: Free the composite render surface

	return -1;
}

HRESULT vEngine::InitDeviceObjects()
{
	if(FAILED(D3DXCreateTextureFromFile(pD3DDevice, "Data/Textures/noise.png", &texNoiseNormals))) return E_FAIL;

	// TODO: Allocate buffers
	/*if(FAILED(pD3DDevice->CreateTexture(windowWidth, windowHeight, 1, 0, 
										D3DFMT_A8R8G8B8, D3DPOOL_DEFAULT, &texAlbedo, 0))) return E_FAIL;
	if(FAILED(pD3DDevice->CreateTexture(windowWidth, windowHeight, 1, 0, 
										D3DFMT_R32F, D3DPOOL_DEFAULT, &texDepth, 0))) return E_FAIL;*/

	if(FAILED(pD3DDevice->CreateTexture(windowWidth/2, windowHeight/2, 1, D3DUSAGE_RENDERTARGET, 
										D3DFMT_R16F, D3DPOOL_DEFAULT, &texSpareIntensity, 0))) return E_FAIL;
	if(FAILED(pD3DDevice->CreateTexture(windowWidth, windowHeight, 1, D3DUSAGE_RENDERTARGET, 
										D3DFMT_A16B16G16R16F, D3DPOOL_DEFAULT, &texNormalDepth, 0))) return E_FAIL;
	if(FAILED(pD3DDevice->CreateTexture(windowWidth/2, windowHeight/2, 1, D3DUSAGE_RENDERTARGET, 
										D3DFMT_A16B16G16R16F, D3DPOOL_DEFAULT, &texNormalDepth4, 0))) return E_FAIL;
	if(FAILED(pD3DDevice->CreateTexture(windowWidth/2, windowHeight/2, 1, D3DUSAGE_RENDERTARGET, 
										D3DFMT_R16F, D3DPOOL_DEFAULT, &texAmbientOcclusion, 0))) return E_FAIL;
	if(FAILED(pD3DDevice->CreateTexture(windowWidth, windowHeight, 1, D3DUSAGE_RENDERTARGET, 
										D3DFMT_A16B16G16R16F, D3DPOOL_DEFAULT, &texLight, 0))) return E_FAIL;

	views.push_back(texLight);
	//views.push_back(texAlbedo);
	views.push_back(texNormalDepth);
	views.push_back(texNormalDepth4);
	views.push_back(texAmbientOcclusion);

	for (unsigned int i = 0; i < views.size(); i++)
	{
		views[i]->AddRef();
	}

	if (!bDeviceLost) 
	{
		// TODO: Register buffers with CUDA
		/*cudaD3D9RegisterResource(texAlbedo, cudaD3D9RegisterFlagsNone);
		cutilCheckMsg("cudaD3D9RegisterResource (g_texture_2d) failed");

		cudaD3D9ResourceSetMapFlags(texAlbedo, cudaD3D9MapFlagsWriteDiscard);
		cutilCheckMsg("cudaD3D9ResourceSetMapFlags (g_texture_2d) failed");

		cudaD3D9RegisterResource(texDepth, cudaD3D9RegisterFlagsNone);
		cutilCheckMsg("cudaD3D9RegisterResource (g_texture_2d) failed");

		cudaD3D9ResourceSetMapFlags(texDepth, cudaD3D9MapFlagsWriteDiscard);
		cutilCheckMsg("cudaD3D9ResourceSetMapFlags (g_texture_2d) failed");*/
	}

	return S_OK;
}

HRESULT vEngine::DestroyDeviceObjects()
{
	// TODO: Unregister buffers with CUDA
	/*cudaD3D9UnregisterResource(texAlbedo);
	cutilCheckMsg("cudaD3D9UnregisterResource (g_texture_2d) failed");

	cudaD3D9UnregisterResource(texDepth);
	cutilCheckMsg("cudaD3D9UnregisterResource (g_texture_2d) failed");*/

	for (unsigned int i = 0; i < views.size(); i++)
	{
		SafeRelease(views[i]);
	}

	views.clear();

	SafeRelease(texNoiseNormals);
	// TODO: Release buffers
	//SafeRelease(texAlbedo);
	//SafeRelease(texDepth);
	SafeRelease(texSpareIntensity);
	SafeRelease(texNormalDepth);
	SafeRelease(texNormalDepth4);
	SafeRelease(texAmbientOcclusion);
	SafeRelease(texLight);

	return S_OK;
}

HRESULT vEngine::DeviceLostHandler()
{
	HRESULT hr;

	fprintf(stderr, "-> Starting DeviceLostHandler() \n");

	if (FAILED(hr = pD3DDevice->TestCooperativeLevel()))
	{
		fprintf(stderr, "TestCooperativeLevel = %08x failed, will attempt to reset\n", hr);

		if (hr == D3DERR_DEVICELOST) 
		{
			fprintf(stderr, "TestCooperativeLevel = %08x DeviceLost, will retry next call\n", hr);

			return S_OK;
		}

		if (hr == D3DERR_DEVICENOTRESET)
		{
			fprintf(stderr, "TestCooperativeLevel = %08x will try to RESET the device\n", hr);

			if (bWindowed)
			{
				pD3D->GetAdapterDisplayMode(D3DADAPTER_DEFAULT, &d3ddm);

				d3dpp.BackBufferFormat = d3ddm.Format;
			}

			if (FAILED(hr = pD3DDevice->Reset(&d3dpp))) 
			{
				fprintf(stderr, "TestCooperativeLevel = %08x RESET device FAILED!\n", hr);

				return hr;
			} 
			else 
			{
				fprintf(stderr, "TestCooperativeLevel = %08x RESET device SUCCESS!\n", hr);

				cudaD3D9SetDirect3DDevice(pD3DDevice);
				cutilCheckMsg("cudaD3D9SetDirect3DDevice failed");

				InitDeviceObjects();

				bDeviceLost = false;
			}
		}
	}

	return hr;
}

CGparameter vEngine::GetUserParam(const std::string &typeName)
{
	ParameterList &params(userParams[typeName]);
	size_t &used(userParamsUsed[typeName]);

	if (used == params.size())
	{
		CGtype type(cgGetNamedUserType(effect, typeName.c_str()));

		params.push_back(cgCreateParameter(context, type));
	}

	return params[used++];
}

void vEngine::ResetUserParams()
{
	for(std::map<std::string,size_t>::iterator it(userParamsUsed.begin()), end(userParamsUsed.end()); it!=end; ++it)
	{
		it->second = 0;
	}
}

void vEngine::SetEffect(const std::string &name)
{
	static const std::string shaderPath("Data/Effects/");

	effectName = name;

	EffectMap::const_iterator it(effects.find(name));

	if (it == effects.end())
	{
		CGeffect effect(cgCreateEffectFromFile(context, (shaderPath + name + ".cgfx").c_str(), 0));

		CheckCgError("cgCreateEffectFromFile(\"" + name + "\")");

		effects.insert(std::make_pair(name, effect));
		this->effect = effect;
	}
	else
	{
		this->effect = it->second;
	}
}

void vEngine::RemoveEffect()
{
	std::cerr << "Warning: Removing effect \"" << effectName << "\" due to Cg error." << std::endl;

	effects[effectName] = 0;
}

CGparameter vEngine::GetParam(const std::string &name)
{
	if (!effect) return 0;

	CGparameter param(cgGetNamedEffectParameter(effect, name.c_str()));

	if (CheckCgError("cgGetNamedEffectParameter(" + effectName + ", " + name + ")"))
	{
		RemoveEffect();

		return 0;
	}
	else if(!param)
	{
		std::cerr << "No parameter \"" + name + "\" in effect \"" + effectName+"\"" << std::endl;

		RemoveEffect();

		return 0;
	}

	return param;
}

void vEngine::SetParam(const std::string &name, float x)
{
	CGparameter param(GetParam(name)); 
	if (!param) return;

	cgSetParameter1f(param, x);

	if (CheckCgError("cgSetParameter1f")) RemoveEffect();
}

void vEngine::SetParam(const std::string &name, float x, float y)
{
	CGparameter param(GetParam(name)); 
	if (!param) return;

	cgSetParameter2f(param, x, y);

	if (CheckCgError("cgSetParameter2f")) RemoveEffect();
}

void vEngine::SetParam(const std::string &name, float x, float y, float z)
{
	CGparameter param(GetParam(name)); 
	if (!param) return;

	cgSetParameter3f(param, x, y, z);

	if (CheckCgError("cgSetParameter3f")) RemoveEffect();
}

void vEngine::SetParam(const std::string &name, float x, float y, float z, float w)
{
	CGparameter param(GetParam(name)); 
	if (!param) return;

	cgSetParameter4f(param, x, y, z, w);

	if (CheckCgError("cgSetParameter4f")) RemoveEffect();
}

void vEngine::SetParam(const std::string &name, bool v)
{
	SetParam(name, v ? 1.0f : 0.0f);
}

void vEngine::SetParam(const std::string &name, IDirect3DTexture9 *tex)
{
	CGparameter param(GetParam(name)); 
	if (!param) return;

	cgD3D9SetTextureParameter(param, tex);

	if (CheckCgError("cgD3D9SetTextureParameter")) RemoveEffect();
}

void vEngine::RenderPass(const std::string &passName, IDirect3DTexture9 *target0, IDirect3DTexture9 *target1)
{
	struct VertexStruct
	{
		float position[3];
		float texture[2];
	};

	int width  = windowWidth;
	int height = windowHeight;

	if (target0)
	{
		D3DSURFACE_DESC dest; target0->GetLevelDesc(0, &dest);

		width  = dest.Width;
		height = dest.Height;
	}

	const float biasS = 0.5f / width;
	const float biasT = 0.5f / height;
	const float s0 = biasS, s1 = biasS + 1.0f;
	const float t0 = biasT, t1 = biasT + 1.0f;

	const VertexStruct vertices[4] = 
	{
		{  {-1,-1,0}, {s0,t0} },
		{  {-1, 1,0}, {s0,t1} },
		{  { 1, 1,0}, {s1,t1} },
		{  { 1,-1,0}, {s1,t0} }
	};

	const VertexStruct flipVertices[4] = 
	{
		{  {-1,-1,0}, {s0,t1} },
		{  {-1, 1,0}, {s0,t0} },
		{  { 1, 1,0}, {s1,t0} },
		{  { 1,-1,0}, {s1,t1} }
	};

	CGtechnique technique(cgGetFirstTechnique(effect));

	while (technique && cgValidateTechnique(technique) == CG_FALSE) 
	{
		technique = cgGetNextTechnique(technique);
	}

	if (technique)
	{
		CGpass pass = cgGetNamedPass(technique, passName.c_str());

		if (pass)
		{
			IDirect3DSurface9 *surface0 = frameBuffer;
			if (target0) target0->GetSurfaceLevel(0, &surface0);

			pD3DDevice->SetRenderTarget(0, surface0);

			if (target1)
			{
				IDirect3DSurface9 *surface1; target1->GetSurfaceLevel(0, &surface1);

				pD3DDevice->SetRenderTarget(1, surface1);
			}

			pD3DDevice->BeginScene();

			cgSetPassState(pass);

			pD3DDevice->SetFVF(D3DFVF_XYZ | D3DFVF_TEX1);
			pD3DDevice->DrawPrimitiveUP(D3DPT_TRIANGLEFAN, 2, target0 ? flipVertices : vertices, sizeof(VertexStruct));

			cgResetPassState(pass);

			pD3DDevice->EndScene();

			if (target1)
			{
				pD3DDevice->SetRenderTarget(1, NULL);
			}
		}
	}		
}

void vEngine::DownSample2x2(IDirect3DTexture9 *texSource, const std::string &pass, IDirect3DTexture9 *texDest)
{
	D3DSURFACE_DESC dest; texDest->GetLevelDesc(0, &dest);

	float texelWidth	= 1.0f / dest.Width;	// How many source texels per dest texel
	float texelHeight	= 1.0f / dest.Height;	// How many source texels per dest texel
	float sampleWidth	= texelWidth / 2.0f;	// How many source texels per dest sample
	float sampleHeight	= texelHeight / 2.0f;	// How many source texels per dest sample

	float offsets[4][2];

	for (int t = 0; t < 2; t++)
	{
		for (int s = 0; s < 2; s++)
		{
			offsets[t * 2 + s][0] = (s * 1.0f - 0.5f) * sampleWidth;
			offsets[t * 2 + s][1] = (t * 1.0f - 0.5f) * sampleHeight;
		}
	}

	SetEffect("filter");
	if (!effect) return;

	SetParam("TexSource", texSource);

	CGparameter param = cgGetNamedEffectParameter(effect, "SampleOffsets2x2");
	cgSetParameterValuefr(param, 8, offsets[0]);

	if(CheckCgError("cgSetParameterValuefr"))
	{
		RemoveEffect();

		return;
	}

	RenderPass(pass, texDest);
}

void vEngine::DownSample4x4(IDirect3DTexture9 *texSource, const std::string &pass, IDirect3DTexture9 *texDest)
{
	D3DSURFACE_DESC dest; texDest->GetLevelDesc(0, &dest);

	float texelWidth	= 1.0f / dest.Width;	// How many source texels per dest texel
	float texelHeight	= 1.0f / dest.Height;	// How many source texels per dest texel
	float sampleWidth	= texelWidth / 4.0f;	// How many source texels per dest sample
	float sampleHeight	= texelHeight / 4.0f;	// How many source texels per dest sample

	float offsets[16][2];

	for (int t = 0; t < 4; t++)
	{
		for (int s = 0; s < 4; s++)
		{
			offsets[t * 4 + s][0] = (s * 1.0f - 1.5f) * sampleWidth;
			offsets[t * 4 + s][1] = (t * 1.0f - 1.5f) * sampleHeight;
		}
	}

	SetEffect("filter");
	if (!effect) return;

	SetParam("TexSource", texSource);

	CGparameter param = cgGetNamedEffectParameter(effect, "SampleOffsets4x4");
	cgSetParameterValuefr(param, 32, offsets[0]);

	if (CheckCgError("cgSetParameterValuefr"))
	{
		RemoveEffect();

		return;
	}

	RenderPass(pass, texDest);
}

void vEngine::RenderSSN()
{
	SetEffect("ssn");
	if (!effect) return;

	// TODO: Fix screenspace normals pass
	//SetParam("TexDepth",        texDepth);
	//SetParam("TexAlbedo",       texAlbedo);
	SetParam("TexelSize",       texelSize[0], texelSize[1]);
	SetParam("FrustumExtents",  frustumExtents[0], frustumExtents[1]);
	SetParam("InverseProjectA", 1.0f / projectionMatrix.i22);
	SetParam("ProjectB",        projectionMatrix.i32);
	
	RenderPass("NormalDepth", texNormalDepth, NULL);
}

void vEngine::RenderSSAO()
{
	if (!passEnabled[PASS_AMBIENT_OCCLUSION]) return;

	DownSample2x2(texNormalDepth, "TapNormal2x2", texNormalDepth4);

	const float depthScaling = 0.1f;

	SetEffect("ssao");
	if (!effect) return;

	SetParam("TexNormalDepth4",	texNormalDepth4);
	SetParam("TexNoiseNormals", texNoiseNormals);
	SetParam("FrustumExtents",	frustumExtents[0], frustumExtents[1]);
	SetParam("NoiseScale",		windowWidth / 64.0f, windowHeight / 64.0f);
	SetParam("DepthScaling",	depthScaling);

	RenderPass("SSAO", texAmbientOcclusion);

	SetEffect("filter");
	if (!effect) return;

	SetParam("TexNormalDepth",	texNormalDepth);
	SetParam("CbDepthScaling",	depthScaling);
	SetParam("TexSource",		texAmbientOcclusion);
	SetParam("CbSampleOffset",	1.0f / windowWidth, 0.0f);

	RenderPass("CrossBilateral", texSpareIntensity);

	SetParam("TexSource",		texSpareIntensity);
	SetParam("CbSampleOffset",	0.0f, 1.0f / windowHeight);

	RenderPass("CrossBilateral", texAmbientOcclusion);
}

void vEngine::RenderLight()
{
	SetEffect("light");
	if (!effect) return;

	// TODO: Fix lighting pass
	SetParam("TexNormalDepth",		   texNormalDepth);
	//SetParam("TexAlbedo",			   texAlbedo);
	SetParam("TexelSize",              texelSize[0], texelSize[1]);
	SetParam("FrustumExtents",		   frustumExtents[0], frustumExtents[1]);
	SetParam("EnableAmbientOcclusion", passEnabled[PASS_AMBIENT_OCCLUSION]);

	if (passEnabled[PASS_AMBIENT_OCCLUSION])
	{
		D3DSURFACE_DESC desc; texAmbientOcclusion->GetLevelDesc(0, &desc);

		SetParam("TexelSize4",           1.0f / desc.Width, 1.0f / desc.Height);
		SetParam("TexNormalDepth4",      texNormalDepth4);
		SetParam("TexAmbientOcclusion4", texAmbientOcclusion);
	}

	SetParam("EnableDirectLighting",	passEnabled[PASS_DIRECT_LIGHTING]);
	SetParam("EnableAlbedoTexture",		passEnabled[PASS_ALBEDO_TEXTURE]);
	SetParam("EnableAmbientFiltering",	passEnabled[PASS_AMBIENT_FILTERING]);

	CGparameter param(cgGetNamedEffectParameter(effect, "Lights"));

	static bool first(true); // TODO: Fix to allow dynamic numbers of light sources

	if (first) cgSetArraySize(param, lights.size());

	if (cgGetError() == CG_NO_ERROR)
	{
		for (unsigned int i = 0; i < lights.size(); i++)
		{
			CGparameter light(GetUserParam(lights[i]->GetTypename()));
			lights[i]->SetParameters(light);

			if (first) cgConnectParameter(light, cgGetArrayParameter(param, i));
		}
	}

	first = false; // TODO: Fix to allow dynamic numbers of light sources

	RenderPass("Light", texLight);
	CheckCgError("RenderPass(texLight)");

	/*
	for (int i = 0; i < lights.size(); i++)
	{
		cgDisconnectParameter(cgGetArrayParameter(param, i));
	}
	*/

	ResetUserParams();
}

int vEngine::CompositeBeginScene(vEngine::RenderTarget * target)
{
	// TODO: Use target

	for (unsigned int i = 0; i < lights.size(); i++)
	{
		delete lights[i];
	}

	lights.clear();

	if (!bDeviceLost) 
	{
		// TODO: Fix mapping phase
		/*IDirect3DResource9 * ppResources[2] = {texAlbedo, texDepth};
		cudaD3D9MapResources(2, ppResources);

		cutilCheckMsg("cudaD3D9MapResources(2) failed");

		cutilSafeCallNoSync(cudaD3D9ResourceGetMappedPointer((void**)&sceneConstants.compositeFBuffer[0], texAlbedo, 0, 0));
		cutilSafeCallNoSync(cudaD3D9ResourceGetMappedPointer((void**)&sceneConstants.compositeZBuffer[0], texDepth, 0, 0));*/

		return 0;
	}
	else
	{
		return -1;
	}
}

int vEngine::CompositeEndScene()
{
	// TODO: Fix unmapping phase
	/*IDirect3DResource9 * ppResources[2] = {texAlbedo, texDepth};
	cudaD3D9UnmapResources(2, ppResources);

	cutilCheckMsg("cudaD3D9UnmapResources(2) failed");*/

	if (bDeviceLost) 
	{
		if (FAILED(DeviceLostHandler()))
		{
			fprintf(stderr, "DeviceLostHandler FAILED!\n");

			return -1;
		}
	}

	if (!bDeviceLost) 
	{
		texelSize[0] = 1.0f / windowWidth;
		texelSize[1] = 1.0f / windowHeight;

		frustumExtents[0] = 1.0f / projectionMatrix.i00;
		frustumExtents[1] = 1.0f / projectionMatrix.i11;
		
		if (view == 0)
		{
			SetEffect("filter");

			if (effect)
			{
				// TODO: Fix pass-through renderer
				// SetParam("TexSource", texAlbedo);
				RenderPass("Tap");

				CheckCgError("RenderPass(\"Tap\")");
			}
		}
		else
		{
			RenderSSN();
			RenderSSAO();
			RenderLight();

			SetEffect("filter");

			if (effect)
			{
				SetParam("TexSource", views[view - 1]);
				RenderPass("Tap");

				CheckCgError("RenderPass(\"Tap\")");
			}
		}

		if (pD3DDevice->Present(0, 0, 0, 0) == D3DERR_DEVICELOST) 
		{
			bDeviceLost = true;

			fprintf(stderr, "DrawScene Present = Detected D3D DeviceLost\n");
			
			DestroyDeviceObjects();

			return -1;
		}
	}

	return 0;
}