#include "vtrace.h"

vEngine *__engine = NULL;

void __cudaSafeCallNoSync(hipError_t error, const char *file, const int line)
{
    if (error != hipSuccess)
	{
        fprintf(stderr, "cudaSafeCallNoSync() Runtime API error in file <%s>, line %i : %s.\n", file, line, hipGetErrorString(error));

        //exit(-1);
    }
}

void __cutilCheckError(CUTBoolean error, const char *file, const int line)
{
    if (error != CUTTrue)
	{
        fprintf(stderr, "CUTIL CUDA error in file <%s>, line %i.\n", file, line);

        //exit(-1);
    }
}

void __cutilCheckMsg(const char *errorMessage, const char *file, const int line)
{
    hipError_t error = hipGetLastError();

    if (error != hipSuccess)
	{
        fprintf(stderr, "cutilCheckMsg() CUTIL CUDA error: %s in file <%s>, line %i : %s.\n", errorMessage, file, line, hipGetErrorString(error));

        //exit(-1);
    }

    error = hipDeviceSynchronize();

    if (error != hipSuccess)
	{
        fprintf(stderr, "cutilCheckMsg hipDeviceSynchronize error: %s in file <%s>, line %i : %s.\n", errorMessage, file, line, hipGetErrorString(error));

        //exit(-1);
    }
}

vEngine::vEngine()
{
}

vEngine::~vEngine()
{
	Shutdown();
}

int vEngine::Init(int width, int height, bool vsync, bool windowed, bool tripleBuffering)
{
	if (__engine != NULL)
	{
		return -1;
	}
	else
	{
		lastTicks   = GetTickCount();
		elapsedTime = 0;
		frames      = 0;

		windowWidth  = width;
		windowHeight = height;

		RECT windowRect; SetRect(&windowRect, 0, 0, width, height);
		if (windowed) AdjustWindowRect(&windowRect, WS_OVERLAPPEDWINDOW, false);

		const int realWindowWidth  = windowRect.right - windowRect.left;
		const int realWindowHeight = windowRect.bottom - windowRect.top;

		const int windowX = windowed ? (GetSystemMetrics(SM_CXFULLSCREEN) - realWindowWidth) / 2 : 0;
		const int windowY = windowed ? (GetSystemMetrics(SM_CYFULLSCREEN) - realWindowHeight) / 2 : 0;

		WNDCLASSEX wc = {sizeof(WNDCLASSEX), CS_CLASSDC, MsgProc, 0L, 0L, GetModuleHandle(0), 0, 0, 0, 0, "VTRACE", 0};
		RegisterClassEx(&wc);

		hWindow = CreateWindow("VTRACE", "VTrace Test", windowed ? WS_OVERLAPPEDWINDOW : (WS_EX_TOPMOST | WS_POPUP), windowX, windowY, realWindowWidth, realWindowHeight, 0, 0, wc.hInstance, 0);

		ShowWindow(hWindow, SW_SHOWDEFAULT);
		UpdateWindow(hWindow);

		ShowCursor(windowed);

#ifndef __DEVICE_EMULATION__

		if (CompositeInit(vsync, windowed, tripleBuffering))
		{
			return -1;
		}

#endif

		if (CoreInit())
		{
			return -1;
		}

		__engine = this;

		if (CreateRenderTarget(&primaryRenderTarget, width, height, VTRACE_RENDER_TARGET_COLOR_BUFFER | VTRACE_RENDER_TARGET_Z_BUFFER))
		{
			return -1;
		}

		return 0;
	}
}

int vEngine::Shutdown()
{
	if (!__engine) return -1;

	if (FreeRenderTarget(&primaryRenderTarget))
	{
		return -1;
	}

	if (CoreShutdown())
	{
		return -1;
	}

#ifndef __DEVICE_EMULATION__

	if (CompositeShutdown())
	{
		return -1;
	}

#endif

	hipDeviceReset();
	cutilCheckMsg("hipDeviceReset failed");

	UnregisterClass("VTRACE", GetModuleHandle(NULL));

	__engine = NULL;

	return 0;
}

int vEngine::CreateRenderTarget(RenderTarget *target, unsigned int width, unsigned int height, unsigned int flags)
{
	target->flags = flags;

#ifndef __DEVICE_EMULATION__

	if (CompositeCreateRenderSurface(&target->assembled, width, height, flags))
	{
		return -1;
	}

#endif

	for (int i = 0; i < VTRACE_NUM_RECONSTRUCTION_LAYERS; i++)
	{
		int layerWidth = width   / (i + 1);
		int layerHeight = height / (i + 1);

		if (CoreCreateRenderSurface(&target->coreSurfaceLevels[i]))
		{
			return -1;
		}

#ifndef __DEVICE_EMULATION__

		if (CompositeCreateRenderSurface(&target->compositeSurfaceLevels[i], layerWidth, layerHeight, flags))
		{
			return -1;
		}

#endif
	}

	return 0;
}

int vEngine::FreeRenderTarget(vEngine::RenderTarget * target)
{
#ifndef __DEVICE_EMULATION__

	if (CompositeFreeRenderSurface(&target->assembled))
	{
		return -1;
	}

#endif

	for (int i = 0; i < VTRACE_NUM_RECONSTRUCTION_LAYERS; i++)
	{
		if (CoreFreeRenderSurface(&target->coreSurfaceLevels[i]))
		{
			return -1;
		}

#ifndef __DEVICE_EMULATION__

		if (CompositeFreeRenderSurface(&target->compositeSurfaceLevels[i]))
		{
			return -1;
		}

#endif
	}

	return 0;
}

int vEngine::BeginScene(RenderTarget * target)
{
	if (target == NULL) target = &primaryRenderTarget;

#ifndef __DEVICE_EMULATION__

	if (CompositeBeginScene(target))
	{
		return -1;
	}

#endif

	if (CoreBeginScene(target))
	{
		return -1;
	}

	return 0;
}

int vEngine::EndScene()
{
	if (CoreEndScene())
	{
		return -1;
	}

#ifndef __DEVICE_EMULATION__

	if (CompositeEndScene())
	{
		return -1;
	}

#endif

	++frames;	// todo: this should get ripped out, it can be a part of the user app

	DWORD ticks(GetTickCount());

	elapsedTime += (ticks - lastTicks);

	if (elapsedTime > 1000)
	{
		char buff[256] = {0};
		sprintf(buff, "VTrace Test - %i\0", frames);

		SetWindowText(hWindow, buff);

		elapsedTime -= 1000;
		frames = 0;
	}

	lastTicks = ticks;

	return 0;
}

bool vEngine::IsActive() const
{
	return __engine != NULL;
}

HWND vEngine::GetWindowHandle() const
{
	return hWindow;
}

LRESULT WINAPI vEngine::MsgProc(HWND hWnd, UINT msg, WPARAM wParam, LPARAM lParam)	// todo: since the setcurrentview functions and pass functions are public this should be part of the app now too
{
    switch(msg)
    {
    case WM_KEYDOWN:

		switch(wParam)
		{
		case VK_ESCAPE:

            PostQuitMessage(0);

			return 0;

		case '0': __engine->SetCurrentView(0); return 0;
		case '1': __engine->SetCurrentView(1); return 0;
		case '2': __engine->SetCurrentView(2); return 0;
		case '3': __engine->SetCurrentView(3); return 0;
		case '4': __engine->SetCurrentView(4); return 0;
		case '5': __engine->SetCurrentView(5); return 0;
		case '6': __engine->SetCurrentView(6); return 0;
		case '7': __engine->SetCurrentView(7); return 0;
		case '8': __engine->SetCurrentView(8); return 0;
		case '9': __engine->SetCurrentView(9); return 0;

		case 'Q': __engine->TogglePass(PASS_AMBIENT_OCCLUSION); return 0;
		case 'W': __engine->TogglePass(PASS_DIRECT_LIGHTING); return 0;
		case 'E': __engine->TogglePass(PASS_ALBEDO_TEXTURE); return 0;
		case 'R': __engine->TogglePass(PASS_AMBIENT_FILTERING); return 0;
		}

		break;

    case WM_DESTROY:

        PostQuitMessage(0);

        return 0;

    case WM_PAINT:

        ValidateRect(hWnd, NULL);

        return 0;
    }

    return DefWindowProc(hWnd, msg, wParam, lParam);
}