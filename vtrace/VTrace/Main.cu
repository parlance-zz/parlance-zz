#include "hip/hip_runtime.h"
#include <cmath>

#include "vtrace.h"

#define VTEST_GFX_WIDTH		1280
#define VTEST_GFX_HEIGHT	900

#define VTEST_VSYNC			false
#define VTEST_WINDOWED		true
#define VTEST_TRIPLEBUFFER	false

#define VTEST_TIME_SMOOTHING 0.9

double smoothTime = 0.0;
double frameTime  = 1000.0 / 60.0;

__int64 lastTicks;
__int64 tickFrequency;

vEngine::Surface	testSurface;
vEngine::Skin		testSkin;

vEngine engine;

void OurInit()
{
	//engine.LoadSurface(&testSurface, "..\\Data Sets\\Kayla's Models\\rider_body_high.vts");
	//engine.LoadSkin(&testSkin, "..\\Data Sets\\Kayla's Models\\rider_body_high.skn");

	//engine.LoadSurface(&testSurface, "..\\Data Sets\\Kayla's Models\\beast_high.vts");
	//engine.LoadSkin(&testSkin, "..\\Data Sets\\Kayla's Models\\beast_high.skn");

	engine.LoadSurface(&testSurface, "..\\Data Sets\\Misc\\tree.vts");
	engine.LoadSkin(&testSkin, "..\\Data Sets\\Misc\\tree.skn");

	//engine.LoadSurface(&testSurface, "..\\Data Sets\\Kayla's Models\\rider_head_high.vts");
	//engine.LoadSkin(&testSkin, "..\\Data Sets\\Kayla's Models\\rider_head_high.skn");

	//engine.LoadSurface(&testSurface, "..\\Data Sets\\Alien Shark\\alienshark_high.vts");
	//engine.LoadSkin(&testSkin, "..\\Data Sets\\Alien Shark\\alienshark_high.skn");

	vMat4x4 projectionMatrix;
	vMatrixPerspectiveFovLH(&projectionMatrix, 3.14159f / 3.0f, float(VTEST_GFX_WIDTH) / VTEST_GFX_HEIGHT, 1.0f, 100.0f);

	engine.SetProjectionMatrix(&projectionMatrix);

	QueryPerformanceFrequency((LARGE_INTEGER*)&tickFrequency);
	QueryPerformanceCounter((LARGE_INTEGER*)&lastTicks);
}

void OurRender()
{
	__int64 newTicks; QueryPerformanceCounter((LARGE_INTEGER*)&newTicks);

	frameTime   = frameTime * VTEST_TIME_SMOOTHING + ((newTicks - lastTicks) / double(tickFrequency)) * 1000.0 * (1.0 - VTEST_TIME_SMOOTHING);
	smoothTime += frameTime;

	lastTicks = newTicks;

	vVec3 cameraPos(sinf(smoothTime * 0.0005f) * 6.0f, sinf(smoothTime * 0.0002f) * 4.0f, cosf(smoothTime * 0.0002f) * 6.0f);
	//vVec3 cameraPos(sinf(smoothTime * 0.005f) * 8.0f, sinf(smoothTime * 0.002f) * 4.0f, cosf(smoothTime * 0.005f) * 8.0f);

	vVec3 target(0.0f, 0.0f, 0.0f);

	vMat4x4 modelMatrix, viewMatrix;
	vMatrixIdentity(&modelMatrix);
	vMatrixLookAt(&viewMatrix, &cameraPos, &target);

	engine.SetModelMatrix(&modelMatrix);
	engine.SetViewMatrix(&viewMatrix);

	vVec3 up(0.4,-1,0.4), down(-0.4,1,-0.4);
	vVec3 white(1.5,1.25,1.0), dark(0.5f,0.5f, 0.75f);
	vVec3 spec(0.25f, 0.2f, 0.1f);
	vVec3 zero(0,0,0);

	vVec3 p0(20,5,15),p1(-20,10,30);
	vVec3 red(1,0.2f,0.2f),blue(0.2f,0.2f,1);

	engine.LightDirectional(&down, &white, &spec);
	engine.LightDirectional(&up, &dark, &zero);
	//engine.LightPoint(&p0, &dark, &zero);
	//engine.LightPoint(&p1, &dark, &zero);
	
	engine.SetSkin(&testSkin);

	const int testSize = 2;

	srand(0);

	for (int x = -testSize; x <= testSize; x++)
	{
		//for (int y = -testSize; y <= testSize; y++)
		for (int y = 0; y <= 0; y++)
		{
			for (int z = -testSize; z <= testSize; z++)
			{
				//if (!(x==0 && y==0 && z==0))
				{
					//vVec3 translation = vVec3(x, y, z);
					vVec3 translation = vVec3(x + (rand() % 100 - 50) * 0.01f, y, z + (rand() % 100 - 50) * 0.01f);
					vMatrixTranslate(&modelMatrix, &translation);

					vMat4x4 rotationMatrix;
					//vVec3 rotation = vVec3(3.141592635f / 2.0f, 0.0f, 0.0f);
					//vVec3 rotation = vVec3(x + ticks * 0.00025f, y + ticks * 0.00025f, z + ticks * 0.00025f);
					float ang = (rand() % 600) * 0.01f;
					vVec3 rotation = vVec3(0, 0, 0);
					vMatrixPitchYawRoll(&rotationMatrix, &rotation);

					vMat4x4 scaleMatrix;

					vVec3 scale = vVec3(0.8f, 0.8f, 0.8f);

					vMatrixScale(&scaleMatrix, &scale);

					vMatrixMultiply(&modelMatrix, &modelMatrix, &rotationMatrix);
					vMatrixMultiply(&modelMatrix, &modelMatrix, &scaleMatrix);

					engine.SetModelMatrix(&modelMatrix);

					engine.RenderSurface(&testSurface);
				}
			}
		}
	}
}

void OurShutdown()
{
	engine.FreeSkin(&testSkin);
	engine.FreeSurface(&testSurface);
}

int main(int argc, char * argv[])
{
	if (engine.Init(VTEST_GFX_WIDTH, VTEST_GFX_HEIGHT, VTEST_VSYNC, VTEST_WINDOWED, VTEST_TRIPLEBUFFER))
	{
		return -1;
	}

	const HWND hWindow = engine.GetWindowHandle();

	OurInit();

    MSG msg; ZeroMemory(&msg, sizeof(msg));

    while (msg.message != WM_QUIT)
    {
        if (PeekMessage(&msg, NULL, 0U, 0U, PM_REMOVE))
        {
            TranslateMessage(&msg);
            DispatchMessage(&msg);
        }
        else if (!engine.BeginScene())
		{
            OurRender();

			engine.EndScene();
		}
    }

	OurShutdown();

	engine.Shutdown();

	return EXIT_SUCCESS;
}

int WINAPI WinMain(HINSTANCE, HINSTANCE, LPSTR szCmdLine, INT)
{
	return main(1, &szCmdLine);
}