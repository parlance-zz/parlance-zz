#include "hip/hip_runtime.h"
// todo: - finish flush
//       - write bucket generation
//       - write bucket compaction
//       - change out hierarchical buffers and use generate screenspace normals and size using the assumption that normals face the viewer, stored packed and swizzled
//       - change out unswizzle to unpack while it unswizzles for easier processing in shaders
//       - test and optimize
//       - add optimized rendering kernel for shadowmap / depth only with scattered tiny directional lights for global illumination, possibly add direct cubemap rendering

#include "vtrace.h"

texture<unsigned int, 1, hipReadModeElementType> dRenderBucketsTex;

texture<float4,  3,  hipReadModeElementType> dSurfaceLatticesTex;
texture<float4,  3,  hipReadModeElementType> dSurfaceLattices0Tex;

__constant__ DeviceSceneState dSceneState;

__device__ float3 axelUnpack(unsigned int axel)
{
	return make_float3(((axel & 0x000003FF)      ) * (1.0f / 1023.0f),
		               ((axel & 0x001FFC00) >> 10) * (1.0f / 2047.0f),
					   ((axel & 0x7FE00000) >> 21) * (1.0f / 1023.0f));
}

__device__ float4 normalUnpack(unsigned int normal)
{
	return make_float4(((normal & 0x000000FF)       ) * (1.0f / 127.5f) - 1.0f,
		               ((normal & 0x0000FF00) >>   8) * (1.0f / 127.5f) - 1.0f,
					   ((normal & 0x00FF0000) >>  16) * (1.0f / 127.5f) - 1.0f,
					   ((normal & 0xFF000000) >>  24) * (1.0f / 127.5f) - 1.0f);
}

__device__ float3 cross(float3 du, float3 dv)
{
	return make_float3(((du.y * dv.z) - (du.z * dv.y)),
		               ((du.z * dv.x) - (du.x * dv.z)),
					   ((du.x * dv.y) - (du.y * dv.x)));
}

__device__ float3 normalize(float3 n)
{
	float il = rsqrtf(n.x * n.x + n.y * n.y + n.z * n.z);

	return make_float3(n.x * il, n.y * il, n.z * il);
}

__global__ void bucketScanKernel(unsigned int numCompactionIndices)
{
   __shared__  unsigned int temp[1024];

    const unsigned short thid   = threadIdx.x;
	const unsigned short thid2  = thid << 1;
	const unsigned short thid21 = thid2 + 1;
	const unsigned short thid22 = thid21 + 1;

	const unsigned int gid  = (blockIdx.x << 10) + thid2;
	const unsigned int gid1 = gid + 1;

    unsigned short offset = 0;

	if (gid < numCompactionIndices)
	{
		temp[thid2]  = dSceneState.compactionIndicesIn[gid];
		temp[thid21] = dSceneState.compactionIndicesIn[gid1];
	}
	else
	{
		temp[thid2] = 0;
		temp[thid21] = 0;
	}

#pragma unroll
    for (unsigned short d = 512; d > 0; d >>= 1)
    {
        __syncthreads();

        if (thid < d)
        {
            unsigned int ai = (thid21 << offset) - 1;
            unsigned int bi = (thid22 << offset) - 1;

            temp[bi] += temp[ai];
        }

        offset++;
    }

    if (thid == 0)
    {
        temp[1023] = 0;
    }   

#pragma unroll
    for (unsigned short d = 1; d < 1024; d <<= 1)
    {
        offset --;

        __syncthreads();

        if (thid < d)
        {
            unsigned int ai = (thid21 << offset) - 1;
            unsigned int bi = (thid22 << offset) - 1;

            unsigned int t = temp[ai];

            temp[ai]  = temp[bi];
            temp[bi] += t;
        }
    }

    __syncthreads();

    dSceneState.compactionIndicesOut[thid2]  = temp[gid];
    dSceneState.compactionIndicesOut[thid21] = temp[gid1];
}

__global__ void bucketGenerationKernel(unsigned int layerNum, unsigned int axelOffset, unsigned int compactionIndicesOffset)
{
	// todo:
}

__global__ void bucketCompactionKernel(unsigned int numCompactionIndices)
{
	// todo:
}

__global__ void bucketRenderingKernel(unsigned int numBuckets) 
{
	if (blockIdx.x >= numBuckets) return;

	const unsigned int bucketInfo   = tex1Dfetch(dRenderBucketsTex, blockIdx.x);

	const unsigned int surfaceIndex = bucketInfo & 0x0000FFFF;
	const unsigned int axelIndex    = ((bucketInfo & 0xFFFF0000) >> 8) + threadIdx.x;

	const unsigned int _axel     = dSceneState.renderSurfaces[surfaceIndex].surfaceData[axelIndex];

	float3 axelLocalPos = axelUnpack(_axel);

	axelLocalPos.z += surfaceIndex;

	float4 axelPos  = tex3D(dSurfaceLatticesTex,  axelLocalPos.x, axelLocalPos.y, axelLocalPos.z);

	float inverseAxelW  = 1.0f / axelPos.w;

	__shared__ float frameX[256];
	__shared__ float frameY[256];

	unsigned int axelDepth = axelPos.z * inverseAxelW * float(0x00FFFFFFFF);

	frameX[threadIdx.x] = (axelPos.x * inverseAxelW + 0.5f) * dSceneState.renderTarget[0].width;
	frameY[threadIdx.x] = (axelPos.y * inverseAxelW + 0.5f) * dSceneState.renderTarget[0].height;

	unsigned short nextAxelIndex = threadIdx.x + (_axel >> 31);	// discontinuity protection is stored in the high bit

	__syncthreads();

	float size = hypotf(frameX[nextAxelIndex] - frameX[threadIdx.x], frameY[nextAxelIndex] - frameY[threadIdx.x]);

	unsigned char layerNum = min(VTRACE_NUM_RECONSTRUCTION_LAYERS, __float2uint_rd(__log2f(size)));

	frameX[threadIdx.x] = __int_as_float(__float_as_int(frameX[threadIdx.x]) - (layerNum << 23));	// like hell I can't shift floats
	frameY[threadIdx.x] = __int_as_float(__float_as_int(frameY[threadIdx.x]) - (layerNum << 23));

	size = __int_as_float(__float_as_int(size) - (layerNum << 23));

	unsigned int _size = min(unsigned int(size * VTRACE_AXEL_SIZE_SCALE), 255) << 24;

	ushort2 pixelCoords = make_ushort2(__float2int_rd(frameX[threadIdx.x]), __float2int_rd(frameY[threadIdx.x]));

	pixelCoords.x = min(pixelCoords.x, dSceneState.renderTarget[layerNum].width - 1);
	pixelCoords.y = min(pixelCoords.y, dSceneState.renderTarget[layerNum].height - 1);
	
	unsigned int frameBufferFragmentPos  = __umul24(pixelCoords.y & 0xFFFC, dSceneState.renderTarget[layerNum].width) +
			                                       (pixelCoords.x & 0x03) + ((pixelCoords.y & 0x03) << 2) + ((pixelCoords.x & 0xFFFC) << 2);

	if (axelDepth < (dSceneState.renderTarget[layerNum].s_zBuffer[frameBufferFragmentPos] & 0x00FFFFFFFF))
	{
		dSceneState.renderTarget[layerNum].s_zBuffer[frameBufferFragmentPos] = (axelDepth | _size);

		const unsigned int _material = dSceneState.renderSurfaces[surfaceIndex].skinData[axelIndex] | dSceneState.renderSurfaces[surfaceIndex].material;

		float4 axelPos0 = tex3D(dSurfaceLattices0Tex, axelLocalPos.x, axelLocalPos.y, axelLocalPos.z);

		uchar2 subPixelPos = make_uchar2((frameX[threadIdx.x] - pixelCoords.x) * 255.0f, (frameY[threadIdx.x] - pixelCoords.y) * 255.0f);

		float inverseAxelW0 = 1.0f / axelPos0.w;

		uchar2 axelVelocity = make_uchar2(fminf(fmaxf((axelPos0.x * inverseAxelW0 + 0.5f) * dSceneState.renderTarget[0].width  - frameX[threadIdx.x], -127.5f), 127.5f) + 127.5f,
		    					          fminf(fmaxf((axelPos0.y * inverseAxelW0 + 0.5f) * dSceneState.renderTarget[0].height - frameY[threadIdx.x], -127.5f), 127.5f) + 127.5f);

		dSceneState.renderTarget[layerNum].s_vBuffer[frameBufferFragmentPos] = (subPixelPos.x << 24) + (subPixelPos.y << 16) + (axelVelocity.x << 8) + (axelVelocity.y);

		dSceneState.renderTarget[layerNum].s_fBuffer[frameBufferFragmentPos] = _material;
	}
}

__global__ void clearSurfaceKernel(unsigned int layerNum, unsigned int bufferSize)
{
	dSceneState.renderTarget[layerNum].s_zBuffer[min(__umul24(blockIdx.x, gridDim.x) + threadIdx.x, bufferSize)] = 0x00FFFFFF;
}

__global__ void unswizzleSurfaceKernel(unsigned int layerNum)
{
	__shared__ unsigned int zBuffer[128];
	__shared__ unsigned int fBuffer[128];
	__shared__ unsigned int vBuffer[128];

	const unsigned int threadId     = (min(threadIdx.z, (dSceneState.renderTarget[layerNum].width >> 4) - 1) << 4)
		                             + min(threadIdx.x,  dSceneState.renderTarget[layerNum].width & 0x03) + (threadIdx.y << 2);

	const unsigned int blockId      = __umul24(blockIdx.y, gridDim.x) + blockIdx.x;

	const unsigned int sourceOffset = threadId + (blockId << 7);

	zBuffer[threadId] = dSceneState.renderTarget[layerNum].s_zBuffer[sourceOffset];
	fBuffer[threadId] = dSceneState.renderTarget[layerNum].s_fBuffer[sourceOffset];
	vBuffer[threadId] = dSceneState.renderTarget[layerNum].s_vBuffer[sourceOffset];

	const unsigned int sharedOffset  = (threadIdx.y << 4) + ((threadIdx.z & 0xFE) << 1) + ((threadIdx.z & 0x01) << 6) + threadIdx.x;

	const unsigned int destOffset    = min((blockIdx.x << 5) + (threadIdx.y << 2) + ((threadIdx.z & 0x01) << 4) + threadIdx.x, dSceneState.renderTarget[layerNum].width - 1)
		                             + __umul24(min((threadIdx.z >> 1) + (blockIdx.y << 2), dSceneState.renderTarget[layerNum].height - 1), dSceneState.renderTarget[layerNum].width);

	__syncthreads();

#ifndef __DEVICE_EMULATION__

	dSceneState.renderTarget[layerNum].c_zBuffer[destOffset] = zBuffer[sharedOffset];
	dSceneState.renderTarget[layerNum].c_fBuffer[destOffset] = fBuffer[sharedOffset];
	dSceneState.renderTarget[layerNum].c_vBuffer[destOffset] = vBuffer[sharedOffset];

#endif
}

void vEngine::CoreFlushBatch()
{
	// todo:

	//unsigned int compactionIndicesOffset = 0;

	for (int i = 0; i < 5; i++)
	{

	}


	/*
	hipStreamSynchronize(deviceStream);

	void *_sceneState = (void*)sceneState;
	hipMemcpyToSymbolAsync(HIP_SYMBOL(dSceneState), _sceneState, sizeof(DeviceSceneState), 0, hipMemcpyHostToDevice, deviceStream);

	int numBlocks = (numLayerVoxels * 4) / 256;

	dim3 dimBlock(256, 1, 1);
	dim3 dimGrid(32, numBlocks >> 5, 1);

	voxelSurfaceKernel<<<dimGrid, dimBlock, 0, deviceStream>>>();
	*/

	numRenderSurfaces = 0;

	for (int i = 0; i < 5; i++)
	{
		numLayerSurfaces[i] = 0;
	}

	numRenderBuckets = 0;
	numCompactionIndices = 0;
}

void vEngine::RenderSurface(Surface *surface)
{
	vVec4 *latticePoints = (vVec4*)&surfaceLattice;

	float minX = (latticePoints[0].x * (1.0f / latticePoints[0].w) + 0.5f) * primaryRenderTarget.coreSurfaceLevels[0].width;
	float minY = (latticePoints[0].y * (1.0f / latticePoints[0].w) + 0.5f) * primaryRenderTarget.coreSurfaceLevels[0].height;
	float minZ = latticePoints[0].z *  (1.0f / latticePoints[0].w);

	float maxX = minX;
	float maxY = minY;
	float maxZ = minZ;

	for (int p = 1; p < 8; p++)
	{
		float cx = (latticePoints[p].x * (1.0f / latticePoints[0].w) + 0.5f) * primaryRenderTarget.coreSurfaceLevels[0].width;
		float cy = (latticePoints[p].y * (1.0f / latticePoints[0].w) + 0.5f) * primaryRenderTarget.coreSurfaceLevels[0].height;
		float cz = latticePoints[p].z *  (1.0f / latticePoints[0].w);

		if (cx < minX) minX = cx; if (cx > maxX) maxX = cx;
		if (cy < minY) minY = cy; if (cy > maxY) maxY = cy;
		if (cz < minZ) minZ = cz; if (cz > maxZ) maxZ = cz;
	}

	if ((maxX < 0.0f) || (minX > primaryRenderTarget.coreSurfaceLevels[0].width) ||
		(maxY < 0.0f) || (minY > primaryRenderTarget.coreSurfaceLevels[0].height) ||
		(maxZ < 0.0f) || (minZ > 1.0f))
	{
		return;
	}
	
	float sizeX = maxX - minX;
	float sizeY = maxY - minY;

	float size = sizeX > sizeY ? sizeX : sizeY;

	int layerNum = int(log2f(size)) - 4;		// todo: this probably needs a fair bit of tweaking

	layerNum = layerNum < 0 ? 0 : layerNum;
	layerNum = layerNum + 5 > surface->mapLevels ? surface->mapLevels - 5 : layerNum;
	
	unsigned int numSurfaceCompactionIndices = 16 << (layerNum << 1);
	unsigned int numSurfaceBuckets = numSurfaceCompactionIndices << 4;	

	if ((numRenderSurfaces == VTRACE_MAX_BATCH_SURFACES) ||
		(numLayerSurfaces[layerNum] == VTRACE_MAX_BATCH_SURFACES_PER_LAYER) ||
		((numRenderBuckets + numSurfaceBuckets) >= VTRACE_MAX_BATCH_BUCKETS) ||
		((numCompactionIndices + numSurfaceCompactionIndices) >= VTRACE_MAX_COMPACTION_INDICES))
	{
		CoreFlushBatch();
	}

	sceneState->renderSurfaces[numRenderSurfaces].surfaceData = (unsigned int *)surface->surfaceData;
	sceneState->renderSurfaces[numRenderSurfaces].normalsData = (unsigned int *)surface->normalsData;

	sceneState->renderSurfaces[numRenderSurfaces].skinData = (unsigned short *)surfaceSkin->skinData;
	sceneState->renderSurfaces[numRenderSurfaces].material = surfaceMaterial;

	sceneState->layerSurfaceIndices[layerNum][numLayerSurfaces[layerNum]] = numRenderSurfaces;

	surfaceLattices[numRenderSurfaces]  = surfaceLattice;
	surfaceLattices0[numRenderSurfaces] = surfaceLattice0;

	numRenderSurfaces++;
	numLayerSurfaces[layerNum]++;

	numRenderBuckets     += numSurfaceBuckets;
	numCompactionIndices += numSurfaceCompactionIndices;
}

void vEngine::SetModelMatrix(vMat4x4 *matrix, vMat4x4 *matrix0)
{
	vMat4x4 worldProjectionMatrix;
	vMatrixMultiply(&worldProjectionMatrix, &viewProjectionMatrix, matrix);

	vMatrixMultiplyVec4(&surfaceLattice.c0, &worldProjectionMatrix, &identityLattice.c0);
	vMatrixMultiplyVec4(&surfaceLattice.c1, &worldProjectionMatrix, &identityLattice.c1);
	vMatrixMultiplyVec4(&surfaceLattice.c2, &worldProjectionMatrix, &identityLattice.c2);
	vMatrixMultiplyVec4(&surfaceLattice.c3, &worldProjectionMatrix, &identityLattice.c3);
	vMatrixMultiplyVec4(&surfaceLattice.c4, &worldProjectionMatrix, &identityLattice.c4);
	vMatrixMultiplyVec4(&surfaceLattice.c5, &worldProjectionMatrix, &identityLattice.c5);
	vMatrixMultiplyVec4(&surfaceLattice.c6, &worldProjectionMatrix, &identityLattice.c6);
	vMatrixMultiplyVec4(&surfaceLattice.c7, &worldProjectionMatrix, &identityLattice.c7);

	vMat4x4 worldProjectionMatrix0;

	if (matrix0)
	{
		vMatrixMultiply(&worldProjectionMatrix0, &viewProjectionMatrix0, matrix0);
	}
	else
	{
		vMatrixMultiply(&worldProjectionMatrix0, &viewProjectionMatrix0, matrix);
	}

	vMatrixMultiplyVec4(&surfaceLattice0.c0, &worldProjectionMatrix0, &identityLattice.c0);
	vMatrixMultiplyVec4(&surfaceLattice0.c1, &worldProjectionMatrix0, &identityLattice.c1);
	vMatrixMultiplyVec4(&surfaceLattice0.c2, &worldProjectionMatrix0, &identityLattice.c2);
	vMatrixMultiplyVec4(&surfaceLattice0.c3, &worldProjectionMatrix0, &identityLattice.c3);
	vMatrixMultiplyVec4(&surfaceLattice0.c4, &worldProjectionMatrix0, &identityLattice.c4);
	vMatrixMultiplyVec4(&surfaceLattice0.c5, &worldProjectionMatrix0, &identityLattice.c5);
	vMatrixMultiplyVec4(&surfaceLattice0.c6, &worldProjectionMatrix0, &identityLattice.c6);
	vMatrixMultiplyVec4(&surfaceLattice0.c7, &worldProjectionMatrix0, &identityLattice.c7);

	modelMatrix  = *matrix;		// todo: sterling needs to know that if lattices are used we simply won't have a model matrix and we shouldn't depend on having one
	modelMatrix0 = *matrix0;
}

void vEngine::SetLattice(vLattice *lattice, vLattice *lattice0)
{
	vMatrixMultiplyVec4(&surfaceLattice.c0, &viewProjectionMatrix, &lattice->c0);
	vMatrixMultiplyVec4(&surfaceLattice.c1, &viewProjectionMatrix, &lattice->c1);
	vMatrixMultiplyVec4(&surfaceLattice.c2, &viewProjectionMatrix, &lattice->c2);
	vMatrixMultiplyVec4(&surfaceLattice.c3, &viewProjectionMatrix, &lattice->c3);
	vMatrixMultiplyVec4(&surfaceLattice.c4, &viewProjectionMatrix, &lattice->c4);
	vMatrixMultiplyVec4(&surfaceLattice.c5, &viewProjectionMatrix, &lattice->c5);
	vMatrixMultiplyVec4(&surfaceLattice.c6, &viewProjectionMatrix, &lattice->c6);
	vMatrixMultiplyVec4(&surfaceLattice.c7, &viewProjectionMatrix, &lattice->c7);

	if (!lattice0) lattice0 = lattice;

	vMatrixMultiplyVec4(&surfaceLattice0.c0, &viewProjectionMatrix0, &lattice0->c0);
	vMatrixMultiplyVec4(&surfaceLattice0.c1, &viewProjectionMatrix0, &lattice0->c1);
	vMatrixMultiplyVec4(&surfaceLattice0.c2, &viewProjectionMatrix0, &lattice0->c2);
	vMatrixMultiplyVec4(&surfaceLattice0.c3, &viewProjectionMatrix0, &lattice0->c3);
	vMatrixMultiplyVec4(&surfaceLattice0.c4, &viewProjectionMatrix0, &lattice0->c4);
	vMatrixMultiplyVec4(&surfaceLattice0.c5, &viewProjectionMatrix0, &lattice0->c5);
	vMatrixMultiplyVec4(&surfaceLattice0.c6, &viewProjectionMatrix0, &lattice0->c6);
	vMatrixMultiplyVec4(&surfaceLattice0.c7, &viewProjectionMatrix0, &lattice0->c7);
}

void vEngine::SetViewMatrix(vMat4x4 *matrix, vMat4x4 *matrix0)
{
	viewMatrix = (*matrix);

	if (matrix0)
	{
		viewMatrix0 = (*matrix0);
	}
	else
	{
		viewMatrix0 = (*matrix);
	}

	vMatrixMultiply(&viewProjectionMatrix,  &projectionMatrix,  &viewMatrix);
	vMatrixMultiply(&viewProjectionMatrix0, &projectionMatrix0, &viewMatrix0);
}

void vEngine::SetProjectionMatrix(vMat4x4 *matrix, vMat4x4 *matrix0)
{
	projectionMatrix = (*matrix);

	if (matrix0)
	{
		projectionMatrix0 = (*matrix0);
	}
	else
	{
		projectionMatrix0 = (*matrix);
	}

	vMatrixMultiply(&viewProjectionMatrix,  &projectionMatrix,  &viewMatrix);
	vMatrixMultiply(&viewProjectionMatrix0, &projectionMatrix0, &viewMatrix0);
}

void vEngine::SetMaterial(float emissive, float specular, float specPow)
{
	surfaceMaterial = (int(emissive * 32.0f) << 27) | (int(specular * 64.0f) << 21) | (int(specPow * 32.0f) << 16);
}

void vEngine::SetSkin(Skin *skin)
{
	surfaceSkin = skin;
}

int vEngine::CoreBeginScene(RenderTarget *target)
{
	memcpy((void*)sceneState->renderTarget, (void*)target->coreSurfaceLevels, sizeof(CoreRenderSurface) * VTRACE_NUM_RECONSTRUCTION_LAYERS);

	void *_renderTarget = (void*)sceneState->renderTarget;
	hipMemcpyToSymbolAsync(HIP_SYMBOL(dSceneState.renderTarget), _renderTarget, sizeof(CoreRenderSurface) * VTRACE_NUM_RECONSTRUCTION_LAYERS, 0, hipMemcpyHostToDevice, deviceStream);

	for (int i = 0; i < VTRACE_NUM_RECONSTRUCTION_LAYERS; i++)
	{
		dim3 dimBlock(sceneState->renderTarget[i].width >> 7, 1, 1);
		dim3 dimGrid(128, 1, 1);

		clearSurfaceKernel<<<dimGrid, dimBlock, 0, deviceStream>>>(i, (sceneState->renderTarget[i].width * sceneState->renderTarget[i].height) - 1);
	}

	numRenderSurfaces = 0;

	for (int i = 0; i < 5; i++)
	{
		numLayerSurfaces[i] = 0;
	}

	numRenderBuckets = 0;
	numCompactionIndices = 0;

	return 0;
}

int vEngine::CoreEndScene()
{
	if (numRenderBuckets > 0)
	{
		CoreFlushBatch();
	}

	dim3 dimBlock(4, 4, 8);

	for (int i = 0; i < VTRACE_NUM_RECONSTRUCTION_LAYERS; i++)
	{
		dim3 dimGrid(sceneState->renderTarget[i].width >> 5, sceneState->renderTarget[i].height >> 2, 1);

		unswizzleSurfaceKernel<<<dimGrid, dimBlock, 0, deviceStream>>>(i);
	}

	hipDeviceSynchronize();

	return 0;
}

int vEngine::CoreInit()
{
	vMatrixIdentity(&viewMatrix);       vMatrixIdentity(&viewMatrix0);
	vMatrixIdentity(&projectionMatrix); vMatrixIdentity(&projectionMatrix0);

	vLatticeIdentity(&surfaceLattice);
	vLatticeIdentity(&surfaceLattice0);

	vLatticeIdentity(&identityLattice);

	hipHostMalloc((void**)&sceneState, sizeof(DeviceSceneState));

	hipStreamCreate(&deviceStream);

	hipHostMalloc((void**)&surfaceLattices,  sizeof(vLattice) * VTRACE_MAX_BATCH_SURFACES);
	hipHostMalloc((void**)&surfaceLattices0, sizeof(vLattice) * VTRACE_MAX_BATCH_SURFACES);

    dSurfaceLatticesTex.addressMode[0] = hipAddressModeClamp;
    dSurfaceLatticesTex.addressMode[1] = hipAddressModeClamp;
    dSurfaceLatticesTex.filterMode = hipFilterModeLinear;
    dSurfaceLatticesTex.normalized = false;

    dSurfaceLattices0Tex.addressMode[0] = hipAddressModeClamp;
    dSurfaceLattices0Tex.addressMode[1] = hipAddressModeClamp;
    dSurfaceLattices0Tex.filterMode = hipFilterModeLinear;
    dSurfaceLattices0Tex.normalized = false;

	hipChannelFormatDesc latticePointDesc = hipCreateChannelDesc<float4>();

	hipMalloc3DArray(&surfaceLatticesArray,  &latticePointDesc, make_hipExtent(2, 2, 2 * VTRACE_MAX_BATCH_SURFACES));
	hipMalloc3DArray(&surfaceLattices0Array, &latticePointDesc, make_hipExtent(2, 2, 2 * VTRACE_MAX_BATCH_SURFACES));

	hipBindTextureToArray(&dSurfaceLatticesTex,  surfaceLatticesArray,  &latticePointDesc);
	hipBindTextureToArray(&dSurfaceLattices0Tex, surfaceLattices0Array, &latticePointDesc);

    dRenderBucketsTex.addressMode[0] = hipAddressModeClamp;
    dRenderBucketsTex.addressMode[1] = hipAddressModeClamp;
    dRenderBucketsTex.filterMode = hipFilterModePoint;
    dRenderBucketsTex.normalized = false;

	hipMalloc((void**)&sceneState->renderBucketsIn,  VTRACE_MAX_BATCH_BUCKETS * sizeof(unsigned int));
	hipMalloc((void**)&sceneState->renderBucketsOut, VTRACE_MAX_BATCH_BUCKETS * sizeof(unsigned int));

	hipChannelFormatDesc renderBucketDesc = hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindUnsigned);
	hipBindTexture(NULL, &dRenderBucketsTex, sceneState->renderBucketsOut, &renderBucketDesc, VTRACE_MAX_BATCH_BUCKETS * sizeof(unsigned int));

	hipMalloc((void**)&sceneState->compactionIndicesIn,  VTRACE_MAX_COMPACTION_INDICES * sizeof(unsigned int));
	hipMalloc((void**)&sceneState->compactionIndicesOut, VTRACE_MAX_COMPACTION_INDICES * sizeof(unsigned int));

	void *_sceneState = (void*)sceneState;
	hipMemcpyToSymbolAsync(HIP_SYMBOL(dSceneState), _sceneState, sizeof(DeviceSceneState), 0, hipMemcpyHostToDevice, deviceStream);

	numRenderSurfaces = 0;

	for (int i = 0; i < 5; i++)
	{
		numLayerSurfaces[i] = 0;
	}

	numRenderBuckets = 0;
	numCompactionIndices = 0;

	return 0;
}

int vEngine::CoreShutdown()
{
	hipDeviceSynchronize();

	hipHostFree(sceneState);

	hipHostFree(surfaceLattices);
	hipHostFree(surfaceLattices0);

	hipFreeArray(surfaceLatticesArray);
	hipFreeArray(surfaceLattices0Array);

	hipFree(sceneState->renderBucketsIn);
	hipFree(sceneState->renderBucketsOut);

	hipFree(sceneState->compactionIndicesIn);
	hipFree(sceneState->compactionIndicesOut);
	
	hipStreamDestroy(deviceStream);

	return 0;
}

int vEngine::CoreCreateRenderSurface(CoreRenderSurface *renderSurface)
{
	hipMalloc((void**)&renderSurface->s_fBuffer,  renderSurface->width * renderSurface->height * sizeof(unsigned int));
	hipMalloc((void**)&renderSurface->s_vBuffer,  renderSurface->width * renderSurface->height * sizeof(unsigned int));
	hipMalloc((void**)&renderSurface->s_zBuffer,  renderSurface->width * renderSurface->height * sizeof(unsigned int));

	return 0;
}

int vEngine::CoreFreeRenderSurface(CoreRenderSurface *renderSurface)
{
	hipFree(renderSurface->s_fBuffer);
	hipFree(renderSurface->s_vBuffer);
	hipFree(renderSurface->s_zBuffer);	

	return 0;
}

int vEngine::LoadSurface(Surface *surface, const char *fileName)
{
	memset((void*)surface, 0, sizeof(Surface));

	FILE *surfaceFile = fopen(fileName, "rb");

	if (!surfaceFile)
	{
		printf("Unable to open surface file : %s\n", fileName);

		return -1;
	}

	SurfaceHeader header;

	fread(&header, sizeof(SurfaceHeader), 1, surfaceFile);

	if (header.magic != 0x03460566)
	{
		printf("Error loading surface file '%s', not a valid surface file.\n", fileName);

		return -1;
	}
	
	int numLayerElements = 4;
	int numTotalElements = 0;

	for (int l = 0; l < header.mapLevels; l++)
	{
		numLayerElements *= 4;
		numTotalElements += numLayerElements;
	}

	surface->mapLevels  = header.mapLevels;
	surface->sizeOfData = numTotalElements;

	char *surfaceData = new char[numTotalElements * sizeof(unsigned int)];	//todo: support loading new normal layers and do asynchronous loading
	fread(surfaceData, 1, numTotalElements * sizeof(unsigned int), surfaceFile);

	hipMalloc((void**)&surface->surfaceData, numTotalElements * sizeof(unsigned int));
	hipMemcpy((void*)surface->surfaceData, (void*)surfaceData, numTotalElements * sizeof(unsigned int), hipMemcpyHostToDevice);

	delete [] surfaceData;

	surface->bucketLevels = header.mapLevels > 6 ? 6 : header.mapLevels;

	numLayerElements = 4;
	numTotalElements = 0;

	for (int l = 0; l < surface->bucketLevels; l++)
	{
		numLayerElements *= 4;
		numTotalElements += numLayerElements;
	}

	char *normalsData = new char[numTotalElements * sizeof(unsigned int)];	//todo: support loading new normal layers and do asynchronous loading
	fread(normalsData, 1, numTotalElements * sizeof(unsigned int), surfaceFile);

	hipMalloc((void**)&surface->normalsData, numTotalElements * sizeof(unsigned int));
	hipMemcpy((void*)surface->normalsData, (void*)normalsData, numTotalElements * sizeof(unsigned int), hipMemcpyHostToDevice);

	delete [] normalsData;

	return 0;
}

void vEngine::FreeSurface(Surface *surface)
{
	hipFree((void*)surface->surfaceData);
	hipFree((void*)surface->normalsData);
}

int vEngine::LoadSkin(Skin *skin, const char *fileName)
{
	memset((void*)skin, 0, sizeof(Skin));

	FILE *skinFile = fopen(fileName, "rb");

	if (!skinFile)
	{
		printf("Unable to open skin file : %s\n", fileName);

		return -1;
	}

	SurfaceHeader header;

	fread(&header, sizeof(SurfaceHeader), 1, skinFile);

	if (header.magic != 0x03460566)
	{
		printf("Error loading skin file '%s', not a valid skin file.\n", fileName);

		return -1;
	}
	
	int numLayerElements = 4;
	int numTotalElements = 0;

	for (int l = 0; l < header.mapLevels; l++)
	{
		numLayerElements *= 4;
		numTotalElements += numLayerElements;
	}

	skin->mapLevels  = header.mapLevels;
	skin->sizeOfData = numTotalElements;

	unsigned short *skinData = new unsigned short[numTotalElements];	//todo: support 32bit skin formats that include material info and do asynchronous loading

	fread(skinData, 1, numTotalElements * sizeof(unsigned short), skinFile);

	hipMalloc((void**)&skin->skinData, numTotalElements * sizeof(unsigned short));
	hipMemcpy((void*)skin->skinData, (void*)skinData, numTotalElements * sizeof(unsigned short), hipMemcpyHostToDevice);

	delete [] skinData;

	return 0;
}

void vEngine::FreeSkin(Skin *skin)
{
	hipFree((void*)skin->skinData);
}